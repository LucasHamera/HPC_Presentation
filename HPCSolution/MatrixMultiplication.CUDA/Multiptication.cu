#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<cmath>

#include "Multiplication.cuh"

const int BLOCK_SIZE = 16;

__global__ void gpu_square_matrix_mult(float *d_a, float *d_b, float *d_result, int n) 
{
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}


__global__ void gpu_matrix_mult(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void set_identity_kernel(
	float *a, 
	int m,
	int n
)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	    if( col < n && row < m) 
		{
			a[row * n + col] = (row == col) ? 1.0f: 0.0f;
		}
} 

	int multipy_1d(
		float  *a,
		float  *b,
		float  *c,
		const int matrix_size
	)
	{    
		
		float *a_gpu;
		float *b_gpu;
		float *c_gpu;
		hipError_t cudaStatus;

		const auto matrix_size_pow = matrix_size * matrix_size;
		const auto matrix_byte_size = matrix_size_pow * sizeof(float);

		cudaStatus = hipMalloc((void**)&a_gpu, matrix_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&b_gpu, matrix_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&c_gpu, matrix_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		    cudaStatus = hipMemcpy(a_gpu, a, matrix_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	    cudaStatus = hipMemcpy(b_gpu, b, matrix_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	    cudaStatus = hipMemcpy(c_gpu, c, matrix_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	    unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu, matrix_size); 

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, c_gpu, matrix_byte_size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}


	
		Error:
		hipFree(a_gpu);
		hipFree(b_gpu);
		hipFree(c_gpu);
    
		return static_cast<int>(cudaStatus);
	}

	int multipy_1d_diff_dim(
		float * a, 
		float * b, 
		float * c, 
		const int m,
		const int n,
		const int k
	)
	{
		float *a_gpu;
		float *b_gpu;
		float *c_gpu;
		hipError_t cudaStatus;

		const int a_gpu_byte_size = sizeof(float)*m*n;
		cudaStatus = hipMalloc((void**)&a_gpu, a_gpu_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		const int b_gpu_byte_size = sizeof(int)*n*k;
		cudaStatus = hipMalloc((void**)&b_gpu, b_gpu_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		const int c_gpu_byte_size = sizeof(int)*n*k;
		cudaStatus = hipMalloc((void**)&c_gpu,  sizeof(int)*m*k);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		    cudaStatus = hipMemcpy(a_gpu, a, a_gpu_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	    cudaStatus = hipMemcpy(b_gpu, b, b_gpu_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	    cudaStatus = hipMemcpy(c_gpu, c, c_gpu_byte_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

		unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		gpu_matrix_mult<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu, m, n, k); 

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, c_gpu, c_gpu_byte_size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	
		Error:
		hipFree(a_gpu);
		hipFree(b_gpu);
		hipFree(c_gpu);
    
		return static_cast<int>(cudaStatus);
	}

	int alocate_1d(
		float** a, 
		const int matrix_size
	)
	{
	
		hipError_t cudaStatus;

		const auto matrix_size_pow = matrix_size * matrix_size;
		const auto matrix_byte_size = matrix_size_pow * sizeof(float);

		cudaStatus = hipMalloc((void**)a, matrix_byte_size);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		Error:
		hipFree(a);

		return static_cast<int>(cudaStatus);
	}

	int free_1d(
		float * a
	)
	{
		hipFree(a);
		return 0;
	}

	void set_identity_1d(
		float * a, 
		const int matrix_size
	)
	{
		hipError_t cudaStatus;

	    unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		set_identity_kernel<<<dimGrid, dimBlock>>>(a, matrix_size, matrix_size); 

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	
		Error:
	}

	int multipy_1d_only_gpu(
		float  *a,
		float  *b,
		float  *c,
		const int matrix_size
	)
	{
		hipError_t cudaStatus;

		const auto matrix_size_pow = matrix_size * matrix_size;
		const auto matrix_byte_size = matrix_size_pow * sizeof(float);

	    unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, matrix_size); 

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		Error:
		return static_cast<int>(cudaStatus);
	}

	int multipy_1d_diff_dim_only_gpu(
		float  *a,
		float  *b,
		float  *c,
		const int m,
		const int n, 
		const int k
	)
	{
		hipError_t cudaStatus;

		unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
		unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
		dim3 dimGrid(grid_cols, grid_rows);
		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

		gpu_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, m, n, k); 

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	
		Error:
		return static_cast<int>(cudaStatus);

	}
